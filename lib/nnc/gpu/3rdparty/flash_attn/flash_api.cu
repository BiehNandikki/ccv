#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2023, Tri Dao.
 ******************************************************************************/

// Include these 2 headers instead of torch/extension.h since we don't need all of the torch headers.
#include <cutlass/numeric_types.h>

#include "src/flash.h"
#include "src/static_switch.h"

void run_mha_fwd(Flash_fwd_params &params, hipStream_t stream, bool force_split_kernel=false) {
    FP16_SWITCH(!params.is_bf16, [&] {
        FWD_HEADDIM_SWITCH(params.d, [&] {
            if (params.num_splits <= 1 && !force_split_kernel) {  // If we don't set it num_splits == 0
                run_mha_fwd_<elem_type, kHeadDim>(params, stream);
            } else {
                run_mha_fwd_splitkv_dispatch<elem_type, kHeadDim>(params, stream);
            }
        });
    });
}

// Find the number of splits that maximizes the occupancy. For example, if we have
// batch * n_heads = 48 and we have 108 SMs, having 2 splits (efficiency = 0.89) is
// better than having 3 splits (efficiency = 0.67). However, we also don't want too many
// splits as that would incur more HBM reads/writes.
// So we find the best efficiency, then find the smallest number of splits that gets 85%
// of the best efficiency.
int num_splits_heuristic(int batch_nheads_mblocks, int num_SMs, int num_n_blocks, int max_splits) {
    // If we have enough to almost fill the SMs, then just use 1 split
    if (batch_nheads_mblocks >= 0.8f * num_SMs) { return 1; }
    max_splits = std::min({max_splits, num_SMs, num_n_blocks});
    float max_efficiency = 0.f;
    std::vector<float> efficiency;
    efficiency.reserve(max_splits);
    auto ceildiv = [](int a, int b) { return (a + b - 1) / b; };
    // Some splits are not eligible. For example, if we have 64 blocks and choose 11 splits,
    // we'll have 6 * 10 + 4 blocks. If we choose 12 splits, we'll have 6 * 11 + (-2) blocks
    // (i.e. it's 11 splits anyway).
    // So we check if the number of blocks per split is the same as the previous num_splits.
    auto is_split_eligible = [&ceildiv, &num_n_blocks](int num_splits) {
        return num_splits == 1 || ceildiv(num_n_blocks, num_splits) != ceildiv(num_n_blocks, num_splits - 1);
    };
    for (int num_splits = 1; num_splits <= max_splits; num_splits++) {
        if (!is_split_eligible(num_splits)) {
            efficiency.push_back(0.f);
        } else {
            float n_waves = float(batch_nheads_mblocks * num_splits) / num_SMs;
            float eff = n_waves / ceil(n_waves);
            // printf("num_splits = %d, eff = %f\n", num_splits, eff);
            if (eff > max_efficiency) { max_efficiency = eff; }
            efficiency.push_back(eff);
        }
    }
    for (int num_splits = 1; num_splits <= max_splits; num_splits++) {
        if (!is_split_eligible(num_splits)) { continue; }
        if (efficiency[num_splits - 1] >= 0.85 * max_efficiency) {
            // printf("num_splits chosen = %d\n", num_splits);
            return num_splits;
        }
    }
    return 1;
}

void run_mha_bwd(Flash_bwd_params &params, hipStream_t stream, const bool configure) {
    FP16_SWITCH(!params.is_bf16, [&] {
        if (params.d <= 32) {
            run_mha_bwd_<elem_type, 32>(params, stream, configure);
        } else if (params.d <= 64) {
            run_mha_bwd_<elem_type, 64>(params, stream, configure);
        } else if (params.d <= 96) {
            run_mha_bwd_<elem_type, 96>(params, stream, configure);
        } else if (params.d <= 128) {
            run_mha_bwd_<elem_type, 128>(params, stream, configure);
        } else if (params.d <= 160) {
            run_mha_bwd_<elem_type, 160>(params, stream, configure);
        } else if (params.d <= 192) {
            run_mha_bwd_<elem_type, 192>(params, stream, configure);
        } else if (params.d <= 224) {
          run_mha_bwd_<elem_type, 224>(params, stream, configure);
        } else if (params.d <= 256) {
          run_mha_bwd_<elem_type, 256>(params, stream, configure);
        }
    });
}
