#include "hip/hip_runtime.h"
extern "C" {
#include <ccv.h>
#include <ccv_internal.h>
#include <nnc/ccv_nnc.h>
#include <nnc/ccv_nnc_easy.h>
#include <nnc/ccv_nnc_internal.h>
}
#include <nnc/gpu/ccv_nnc_compat.h>
#include <hiprand/hiprand_kernel.h>

template<typename NUM>
__global__ void _ccv_nnc_random_normal_kernel_x4(const int count, const uint32_t seed, const float std, const float mean, NUM* const a)
{
	const int id = blockIdx.x * blockDim.x + threadIdx.x;
	hiprandStatePhilox4_32_10_t state;
	hiprand_init(seed, id, 0, &state);
	CUDA_1D_KERNEL_LOOP(i, count) {
		const float4 r = hiprand_normal4(&state); // This is standard normal distribution.
		a[i * 4] = r.x * std + mean;
		a[i * 4 + 1] = r.y * std + mean;
		a[i * 4 + 2] = r.z * std + mean;
		a[i * 4 + 3] = r.w * std + mean;
	}
}

template<typename NUM>
__global__ void _ccv_nnc_random_normal_kernel(const int count, const uint32_t seed, const float std, const float mean, NUM* const a)
{
	const int id = blockIdx.x * blockDim.x + threadIdx.x;
	hiprandStatePhilox4_32_10_t state;
	hiprand_init(seed, id, 0, &state);
	CUDA_1D_KERNEL_LOOP(i, count) {
		const float r = hiprand_normal(&state); // This is standard normal distribution.
		a[i] = r * std + mean;
	}
}

static int _ccv_nnc_random_normal(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(output_size == 1);
	ccv_nnc_tensor_t* const a = outputs[0];
	assert(CCV_IS_TENSOR_CONTIGUOUS(a));
	const int count = ccv_nnc_tensor_count(a->info);
	const uint32_t seed = ccv_nnc_stream_context_genrand_uint32(stream_context);
	const float std = cmd.info.blas.a[0];
	const float mean = cmd.info.blas.a[1];
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	if (count % 4 == 0)
	{
		const int count_4 = count / 4;
		if (a->info.datatype == CCV_32F)
			_ccv_nnc_random_normal_kernel_x4<<<CUDA_GET_BLOCKS(count_4), CUDA_NUM_THREADS, 0, stream>>>(count_4, seed, std, mean, a->data.f32);
		else if (a->info.datatype == CCV_16F)
			_ccv_nnc_random_normal_kernel_x4<<<CUDA_GET_BLOCKS(count_4), CUDA_NUM_THREADS, 0, stream>>>(count_4, seed, std, mean, (__half*)a->data.f16);
	} else {
		if (a->info.datatype == CCV_32F)
			_ccv_nnc_random_normal_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, seed, std, mean, a->data.f32);
		else if (a->info.datatype == CCV_16F)
			_ccv_nnc_random_normal_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, seed, std, mean, (__half*)a->data.f16);
	}
	return CCV_NNC_EXEC_SUCCESS;
}

REGISTER_COMMAND_BACKEND(CCV_NNC_RANDOM_NORMAL_FORWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
	registry->tensor_formats = CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_random_normal;
}

REGISTER_COMMAND_BACKEND(CCV_NNC_RANDOM_NORMAL_BACKWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
	registry->tensor_formats = CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_random_normal;
}
