#include "hip/hip_runtime.h"
extern "C" {
#include <ccv.h>
#include <ccv_internal.h>
#include <nnc/ccv_nnc.h>
#include <nnc/ccv_nnc_easy.h>
#include <nnc/ccv_nnc_internal.h>
}
#include <nnc/gpu/ccv_nnc_compat.h>

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_smooth_l1_forw_kernel(const int batch_size, const int count, const NUM1* const a, const int astep, const NUM2* const b, const int bstep, NUM1* const c, const int cstep, const float beta)
{
	const float beta_inv_2 = 0.5 / beta;
	const float beta_2 = 0.5 * beta;
	CUDA_1D_KERNEL_LOOP(i, batch_size) {
		const NUM1* const ap = a + i * astep;
		const NUM2* const bp = b + i * bstep;
		float p = 0;
		for (int j = 0; j < count; j++)
			p += fabs((float)bp[j] - (float)ap[j]);
		if (p < beta)
		{
			p = 0;
			for (int j = 0; j < count; j++)
				p += ((float)bp[j] - (float)ap[j]) * ((float)bp[j] - (float)ap[j]);
			p *= beta_inv_2;
		} else
			p -= beta_2;
		c[i * cstep] = (NUM1)p;
	}
}

static int _ccv_nnc_smooth_l1_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 2);
	const ccv_nnc_tensor_view_t* a = (ccv_nnc_tensor_view_t*)inputs[0];
	assert(ccv_nnc_tensor_nd(a->info.dim) <= 2);
	const ccv_nnc_tensor_view_t* b = (ccv_nnc_tensor_view_t*)inputs[1];
	assert(output_size == 1);
	ccv_nnc_tensor_view_t* c = (ccv_nnc_tensor_view_t*)outputs[0];
	int dim[CCV_NNC_MAX_DIM_ALLOC];
	int astride[CCV_NNC_MAX_DIM_ALLOC];
	int bstride[CCV_NNC_MAX_DIM_ALLOC];
	int cstride[CCV_NNC_MAX_DIM_ALLOC];
	ccv_nnc_tensor_view_get_dim(a, dim);
	assert(ccv_nnc_tensor_view_check_dim(b, dim));
	ccv_nnc_tensor_view_get_stride(a, astride);
	ccv_nnc_tensor_view_get_stride(b, bstride);
	ccv_nnc_tensor_view_get_stride(c, cstride);
	assert(ccv_nnc_tensor_nd(a->info.dim) <= 2);
	const int batch_size = dim[CCV_NNC_MAX_DIM];
	assert(ccv_nnc_tensor_count(c->info) == batch_size);
	const int count = dim[CCV_NNC_MAX_DIM + 1];
	const int astep = astride[CCV_NNC_MAX_DIM];
	const int bstep = bstride[CCV_NNC_MAX_DIM];
	const int cstep = ccv_nnc_tensor_nd(c->info.dim) == 1 ? 1 : cstride[CCV_NNC_MAX_DIM];
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	const float beta = cmd.info.smooth_l1.beta;
	assert(a->info.datatype == c->info.datatype);
	if (b->info.datatype == CCV_32F)
	{
		if (a->info.datatype == CCV_16F)
			_ccv_nnc_smooth_l1_forw_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, (__half*)a->data.f16, astep, b->data.f32, bstep, (__half*)c->data.f16, cstep, beta);
		else
			_ccv_nnc_smooth_l1_forw_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, a->data.f32, astep, b->data.f32, bstep, c->data.f32, cstep, beta);
	} else {
		assert(b->info.datatype == CCV_16F);
		assert(a->info.datatype == CCV_16F);
		_ccv_nnc_smooth_l1_forw_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, (__half*)a->data.f16, astep, (__half*)b->data.f16, bstep, (__half*)c->data.f16, cstep, beta);
	}
	return CCV_NNC_EXEC_SUCCESS;
}

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_smooth_l1_back_kernel(const int batch_size, const int count, const NUM2* const g, const int gstep, const NUM2* const a, const int astep, const NUM1* const b, const int bstep, const NUM2* const c, const int cstep, NUM2* const h, const int hstep, const float beta)
{
	const float beta_2 = 0.5 * beta;
	const float inv_beta = 1.0 / beta;
	CUDA_1D_KERNEL_LOOP(i, batch_size) {
		const NUM2* const ap = a + i * astep;
		const NUM1* const bp = b + i * bstep;
		NUM2* const hp = h + i * hstep;
		const float cp = (float)c[i * cstep];
		if (cp < beta_2)
		{
			const float gp = inv_beta * (float)g[i * gstep];
			for (int j = 0; j < count; j++)
			{
				const float av = ap[j];
				const float bv = bp[j];
				hp[j] = (NUM2)(gp * (av - bv));
			}
		} else {
			const float gp = (float)g[i * gstep];
			for (int j = 0; j < count; j++)
			{
				const float av = ap[j];
				const float bv = bp[j];
				hp[j] = (NUM2)(((av - bv) > 0 ? 1 : -1) * gp);
			}
		}
	}
}

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_smooth_l1_back_kernel(const int batch_size, const int count, const NUM2* const a, const int astep, const NUM1* const b, const int bstep, const NUM2* const c, const int cstep, NUM2* const h, const int hstep, const float beta)
{
	const float beta_2 = 0.5 * beta;
	const float inv_beta = 1.0 / beta;
	CUDA_1D_KERNEL_LOOP(i, batch_size) {
		const NUM2* const ap = a + i * astep;
		const NUM1* const bp = b + i * bstep;
		NUM2* const hp = h + i * hstep;
		const float cp = (float)c[i * cstep];
		if (cp < beta_2)
			for (int j = 0; j < count; j++)
			{
				const float av = ap[j];
				const float bv = bp[j];
				hp[j] = (NUM2)(inv_beta * (av - bv));
			}
		else
			for (int j = 0; j < count; j++)
			{
				const float av = ap[j];
				const float bv = bp[j];
				hp[j] = (NUM2)((av - bv) > 0 ? 1 : -1);
			}
	}
}

static int _ccv_nnc_smooth_l1_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size >= 3);
	assert(output_size >= 1);
	const ccv_nnc_tensor_view_t* const g = (ccv_nnc_tensor_view_t*)inputs[0];
	assert(!g || !CCV_IS_TENSOR_VIEW(g));
	const ccv_nnc_tensor_view_t* const a = (ccv_nnc_tensor_view_t*)inputs[1];
	const ccv_nnc_tensor_view_t* const b = (ccv_nnc_tensor_view_t*)inputs[2];
	const ccv_nnc_tensor_view_t* const c = (ccv_nnc_tensor_view_t*)inputs[3];
	ccv_nnc_tensor_view_t* const h = (ccv_nnc_tensor_view_t*)outputs[0];
	int dim[CCV_NNC_MAX_DIM_ALLOC];
	int astride[CCV_NNC_MAX_DIM_ALLOC];
	int bstride[CCV_NNC_MAX_DIM_ALLOC];
	int cstride[CCV_NNC_MAX_DIM_ALLOC];
	int hstride[CCV_NNC_MAX_DIM_ALLOC];
	ccv_nnc_tensor_view_get_dim(a, dim);
	assert(ccv_nnc_tensor_view_check_dim(b, dim));
	assert(ccv_nnc_tensor_view_check_dim(h, dim));
	ccv_nnc_tensor_view_get_stride(a, astride);
	ccv_nnc_tensor_view_get_stride(b, bstride);
	ccv_nnc_tensor_view_get_stride(c, cstride);
	ccv_nnc_tensor_view_get_stride(h, hstride);
	assert(ccv_nnc_tensor_nd(a->info.dim) <= 2);
	const int batch_size = dim[CCV_NNC_MAX_DIM];
	assert(ccv_nnc_tensor_count(c->info) == batch_size);
	const int count = dim[CCV_NNC_MAX_DIM + 1];
	const int astep = astride[CCV_NNC_MAX_DIM];
	const int bstep = bstride[CCV_NNC_MAX_DIM];
	const int hstep = hstride[CCV_NNC_MAX_DIM];
	const int cstep = ccv_nnc_tensor_nd(c->info.dim) == 1 ? 1 : cstride[CCV_NNC_MAX_DIM];
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	assert(a->info.datatype == h->info.datatype);
	assert(a->info.datatype == c->info.datatype);
	const int datatype = a->info.datatype;
	const float beta = cmd.info.smooth_l1.beta;
	if (g)
	{
		int gstride[CCV_NNC_MAX_DIM_ALLOC];
		ccv_nnc_tensor_view_get_stride(g, gstride);
		assert(ccv_nnc_tensor_count(g->info) == batch_size);
		const int gstep = ccv_nnc_tensor_nd(g->info.dim) == 1 ? 1 : gstride[CCV_NNC_MAX_DIM];
		assert(g->info.datatype == datatype);
		if (b->info.datatype == CCV_32F)
		{
			if (datatype == CCV_16F)
				_ccv_nnc_smooth_l1_back_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, (__half*)g->data.f16, gstep, (__half*)a->data.f16, astep, b->data.f32, bstep, (__half*)c->data.f16, cstep, (__half*)h->data.f16, hstep, beta);
			else
				_ccv_nnc_smooth_l1_back_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, g->data.f32, gstep, a->data.f32, astep, b->data.f32, bstep, c->data.f32, cstep, h->data.f32, hstep, beta);
		} else {
			assert(b->info.datatype == CCV_16F);
			assert(datatype == CCV_16F);
			_ccv_nnc_smooth_l1_back_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, (__half*)g->data.f16, gstep, (__half*)a->data.f16, astep, (__half*)b->data.f16, bstep, (__half*)c->data.f16, cstep, (__half*)h->data.f16, hstep, beta);
		}
	} else {
		if (b->info.datatype == CCV_32F)
		{
			if (datatype == CCV_16F)
				_ccv_nnc_smooth_l1_back_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, (__half*)a->data.f16, astep, b->data.f32, bstep, (__half*)c->data.f16, cstep, (__half*)h->data.f16, hstep, beta);
			else
				_ccv_nnc_smooth_l1_back_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, a->data.f32, astep, b->data.f32, bstep, c->data.f32, cstep, h->data.f32, hstep, beta);
		} else {
			assert(b->info.datatype == CCV_16F);
			assert(datatype == CCV_16F);
			_ccv_nnc_smooth_l1_back_kernel<<<CUDA_GET_BLOCKS(batch_size), CUDA_NUM_THREADS, 0, stream>>>(batch_size, count, (__half*)a->data.f16, astep, (__half*)b->data.f16, bstep, (__half*)c->data.f16, cstep, (__half*)h->data.f16, hstep, beta);
		}
	}
	return CCV_NNC_EXEC_SUCCESS;
}

REGISTER_COMMAND_BACKEND(CCV_NNC_SMOOTH_L1_FORWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_smooth_l1_forw;
}

REGISTER_COMMAND_BACKEND(CCV_NNC_SMOOTH_L1_BACKWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_smooth_l1_back;
}
