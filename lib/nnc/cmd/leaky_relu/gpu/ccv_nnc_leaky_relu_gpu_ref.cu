#include "hip/hip_runtime.h"
extern "C" {
#include <ccv.h>
#include <ccv_internal.h>
#include <nnc/ccv_nnc.h>
#include <nnc/ccv_nnc_easy.h>
#include <nnc/ccv_nnc_internal.h>
}
#include <nnc/gpu/ccv_nnc_compat.h>

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_leaky_relu_forw_kernel(const size_t count, const float negative_slope, const NUM1* const a, NUM2* const b)
{
	CUDA_1D_KERNEL_LOOP(i, count) {
		float x = (float)a[i];
		b[i] = (NUM2)(x >= 0 ? x : x * negative_slope);
	}
}

static int _ccv_nnc_leaky_relu_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 1);
	const ccv_nnc_tensor_t* const a = inputs[0];
	assert(CCV_IS_TENSOR_CONTIGUOUS(a));
	assert(output_size == 1);
	ccv_nnc_tensor_t* const b = outputs[0];
	assert(CCV_IS_TENSOR_CONTIGUOUS(b));
	const size_t count = ccv_nnc_tensor_count(a->info);
	int i;
	for (i = 0; i < CCV_NNC_MAX_DIM_ALLOC && a->info.dim[i] > 0; i++)
	{
		assert(a->info.dim[i] == b->info.dim[i]);
	}
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	const float negative_slope = cmd.info.leaky_relu.negative_slope;
	if (a->info.datatype == CCV_32F && b->info.datatype == CCV_32F)
	{
		_ccv_nnc_leaky_relu_forw_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, negative_slope, a->data.f32, b->data.f32);
	} else if (a->info.datatype == CCV_32F && b->info.datatype == CCV_16F) {
		_ccv_nnc_leaky_relu_forw_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, negative_slope, a->data.f32, (__half*)b->data.f16);
	} else if (a->info.datatype == CCV_16F && b->info.datatype == CCV_32F) {
		_ccv_nnc_leaky_relu_forw_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, negative_slope, (__half*)a->data.f16, b->data.f32);
	} else if (a->info.datatype == CCV_16F && b->info.datatype == CCV_16F) {
		_ccv_nnc_leaky_relu_forw_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, negative_slope, (__half*)a->data.f16, (__half*)b->data.f16);
	}
	return CCV_NNC_EXEC_SUCCESS;
}

template<typename NUM1, typename NUM2>
__global__ void _ccv_nnc_leaky_relu_back_kernel(const size_t count, const float negative_slope, const NUM1* const b, const NUM2* const g, NUM1* const h)
{
	CUDA_1D_KERNEL_LOOP(i, count) {
		const float x = (float)b[i];
		h[i] = x >= 0 ? (NUM1)g[i] : (NUM1)((float)g[i] * negative_slope);
	}
}

static int _ccv_nnc_leaky_relu_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 3);
	const ccv_nnc_tensor_t* const g = inputs[0]; // gradient
	assert(CCV_IS_TENSOR_CONTIGUOUS(g));
	const ccv_nnc_tensor_t* const b = inputs[2];
	assert(CCV_IS_TENSOR_CONTIGUOUS(b));
	assert(output_size == 1);
	ccv_nnc_tensor_t* const h = outputs[0];
	assert(CCV_IS_TENSOR_CONTIGUOUS(h));
	const size_t count = ccv_nnc_tensor_count(g->info);
	int i;
	for (i = 0; i < CCV_NNC_MAX_DIM_ALLOC && g->info.dim[i] > 0; i++)
	{
		assert(b->info.dim[i] == g->info.dim[i]);
		assert(g->info.dim[i] == h->info.dim[i]);
	}
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	assert(b->info.datatype == h->info.datatype);
	const float negative_slope = cmd.info.leaky_relu.negative_slope;
	if (b->info.datatype == CCV_32F && g->info.datatype == CCV_32F)
	{
		_ccv_nnc_leaky_relu_back_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, negative_slope, b->data.f32, g->data.f32, h->data.f32);
	} else if (b->info.datatype == CCV_32F && g->info.datatype == CCV_16F) {
		_ccv_nnc_leaky_relu_back_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, negative_slope, b->data.f32, (__half*)g->data.f16, h->data.f32);
	} else if (b->info.datatype == CCV_16F && g->info.datatype == CCV_32F) {
		_ccv_nnc_leaky_relu_back_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, negative_slope, (__half*)b->data.f16, g->data.f32, (__half*)h->data.f16);
	} else if (b->info.datatype == CCV_16F && g->info.datatype == CCV_16F) {
		_ccv_nnc_leaky_relu_back_kernel<<<CUDA_GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(count, negative_slope, (__half*)b->data.f16, (__half*)g->data.f16, (__half*)h->data.f16);
	}
	return CCV_NNC_EXEC_SUCCESS;
}

REGISTER_COMMAND_BACKEND(CCV_NNC_LEAKY_RELU_FORWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_leaky_relu_forw;
}

REGISTER_COMMAND_BACKEND(CCV_NNC_LEAKY_RELU_BACKWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_leaky_relu_back;
}
