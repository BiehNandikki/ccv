extern "C" {
#include <ccv.h>
#include <ccv_internal.h>
#include <nnc/ccv_nnc.h>
#include <nnc/ccv_nnc_easy.h>
#include <nnc/ccv_nnc_internal.h>
}
#include <nnc/gpu/ccv_nnc_compat.h>

#ifdef HAVE_CUDA_SM80
#include <nnc/gpu/3rdparty/flash_attn/flash_api.h>

static int _ccv_nnc_scaled_dot_product_attention_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	// NNC notation:
	// C = sm(Q * K^T) * V
	//
	// MFA notation:
	// O = sm(Q * K^T) * V
	assert(input_size >= 3);
	assert(output_size >= 1);
	ccv_nnc_tensor_view_t* const q = (ccv_nnc_tensor_view_t*)inputs[0];
	ccv_nnc_tensor_view_t* const k = (ccv_nnc_tensor_view_t*)inputs[1];
	ccv_nnc_tensor_view_t* const v = (ccv_nnc_tensor_view_t*)inputs[2];
	ccv_nnc_tensor_view_t* const attn_mask = input_size > 3 ? (ccv_nnc_tensor_view_t*)inputs[3] : 0;
	ccv_nnc_tensor_view_t* const weights = input_size > 4 ? (ccv_nnc_tensor_view_t*)inputs[4] : 0;
	ccv_nnc_tensor_view_t* const bias = input_size > 5 ? (ccv_nnc_tensor_view_t*)inputs[5] : 0;
	if (bias) // bias always requires a weight matrix.
		{ assert(weights); }

	ccv_nnc_tensor_view_t* const saved_softmax = NULL;
	ccv_nnc_tensor_view_t* const o = (weights) ? (ccv_nnc_tensor_view_t*)outputs[2] : (ccv_nnc_tensor_view_t*)outputs[0];
	const int q_nd = ccv_nnc_tensor_nd(q->info.dim);
	assert(q_nd == 3 || q_nd == 4);
	const int k_nd = ccv_nnc_tensor_nd(k->info.dim);
	assert(k_nd == 3 || k_nd == 4);
	const int v_nd = ccv_nnc_tensor_nd(v->info.dim);
	assert(v_nd == 3 || v_nd == 4);
	const int o_nd = ccv_nnc_tensor_nd(o->info.dim);
	assert(o_nd == 3 || o_nd == 4);
	assert(q_nd == k_nd && k_nd == v_nd && v_nd == o_nd);

	if (saved_softmax) {
		// MFA does not support a backward pass and cannot store the intermediate
		// softmax. If this is required, fall back to MPSGraph (if will never occur
		// during inference).
		assert(false);
		return CCV_NNC_EXEC_INVALID;
	}

	int qdim[CCV_NNC_MAX_DIM_ALLOC];
	int kdim[CCV_NNC_MAX_DIM_ALLOC];
	int vdim[CCV_NNC_MAX_DIM_ALLOC];
	int odim[CCV_NNC_MAX_DIM_ALLOC];
	int amdim[CCV_NNC_MAX_DIM_ALLOC];
	ccv_nnc_tensor_view_get_dim(q, qdim);
	ccv_nnc_tensor_view_get_dim(k, kdim);
	ccv_nnc_tensor_view_get_dim(v, vdim);
	ccv_nnc_tensor_view_get_dim(o, odim);

	assert(q->info.format == CCV_TENSOR_FORMAT_NHWC);
	assert(k->info.format == CCV_TENSOR_FORMAT_NHWC);
	assert(v->info.format == CCV_TENSOR_FORMAT_NHWC);
	assert(o->info.format == CCV_TENSOR_FORMAT_NHWC);
	if (attn_mask) {
		// MFA does not support fused transposes on the mask.
		assert(attn_mask->info.format == CCV_TENSOR_FORMAT_NHWC);
	}

	assert(CCV_IS_TENSOR_CONTIGUOUS(q));
	assert(CCV_IS_TENSOR_CONTIGUOUS(k));
	assert(CCV_IS_TENSOR_CONTIGUOUS(v));
	assert(CCV_IS_TENSOR_CONTIGUOUS(o));

	if (attn_mask) {
		assert(CCV_IS_TENSOR_CONTIGUOUS(attn_mask));
	}

	int batch_size;
	int R;
	int C;
	int Hq;
	int Hk;
	int D;
	if (q_nd == 3) {
		batch_size = qdim[1];
		assert(batch_size == kdim[1]);
		R = qdim[2];
		C = kdim[2];
		Hq = Hk = 1;
		D = qdim[3];
		assert(D == kdim[3]);
	} else if (q_nd == 4) {
		batch_size = qdim[0];
		assert(batch_size == kdim[0]);
		R = qdim[1];
		C = kdim[1];
		Hq = qdim[2];
		Hk = kdim[2];
		assert(Hq >= Hk);
		assert(Hq % Hk == 0);
		D = qdim[3];
		assert(D == kdim[3]);
	}

	if (attn_mask) {
		// MFA can support am_nd == 2 and broadcast batch=1 -> batch=batch_size, but
		// wait until that occurs in practice before doing so.
		const int am_nd = ccv_nnc_tensor_nd(attn_mask->info.dim);
		assert(am_nd == 3 || am_nd == 4); // [batch_size, R, C]

		// MFA does not support attention mask broadcasting (where the R dimension
		// of Q > 1, but the R dimension of the mask == 1).
		ccv_nnc_tensor_view_get_dim(attn_mask, amdim);
		if (am_nd == 3)
		{
			assert(amdim[1] == batch_size || amdim[1] == 1);
			amdim[0] = amdim[1];
			amdim[1] = 1;
			assert(amdim[2] == R);
			assert(amdim[3] == C);
		} else {
			assert(amdim[0] == batch_size || amdim[0] == 1);
			assert(amdim[1] == 1);
			assert(amdim[2] == R);
			assert(amdim[3] == C);
		}
	}
	int weights_datatype = 0;
	if (weights)
		weights_datatype = CCV_GET_DATA_TYPE(weights->info.datatype) == CCV_QX ? ((weights->info.datatype & 0xff) << 12) : weights->info.datatype;

	const int is_same_dtype =
		(q->info.datatype == k->info.datatype) &&
		(q->info.datatype == v->info.datatype) &&
		(q->info.datatype == o->info.datatype) &&
		(weights ? (q->info.datatype == weights_datatype) : 1) &&
		(bias ? (q->info.datatype == bias->info.datatype) : 1);

	assert(is_same_dtype);

	Flash_fwd_params params;
	memset(&params, 0, sizeof(params));
	params.is_bf16 = false;
	params.q_ptr = q->data.u8;
	params.k_ptr = k->data.u8;
	params.v_ptr = v->data.u8;
	params.q_row_stride = D * Hq;
	params.k_row_stride = D * Hk;
	params.v_row_stride = D * Hk;
	params.q_head_stride = D;
	params.k_head_stride = D;
	params.v_head_stride = D;
	params.q_batch_stride = R * Hq * D;
	params.k_batch_stride = C * Hk * D;
	params.v_batch_stride = C * Hk * D;
	auto round_multiple = [](int x, int m) { return (x + m - 1) / m * m; };
	params.seqlen_q = R;
	params.seqlen_q_rounded = round_multiple(R, 128);
	params.seqlen_k = C;
	params.seqlen_k_rounded = round_multiple(C, 128);
	params.d = D;
	assert(D % 8 == 0);
	params.d_rounded = round_multiple(D, 32);
	params.o_ptr = o->data.u8;
	params.o_row_stride = D * Hq;
	params.o_head_stride = D;
	params.o_batch_stride = R * Hq * D;
	params.b = batch_size;
	params.h = Hq;
	params.h_k = Hk;
	params.h_h_k_ratio = Hq / Hk;
	params.scale_softmax = cmd.info.scaled_dot_product_attention.scale;
	params.scale_softmax_log2 = cmd.info.scaled_dot_product_attention.scale * M_LOG2E;
	params.is_causal = cmd.info.scaled_dot_product_attention.is_causal;
	params.p_dropout = 1;
	params.p_dropout_in_uint8_t = 255;
	params.rp_dropout = 1;
	params.scale_softmax_rp_dropout = params.scale_softmax;
	params.window_size_left = ccv_max(R, C);
	params.window_size_right = params.is_causal ? 0 : ccv_max(R, C);
	params.is_seqlens_k_cumulative = true;
	// TODO: Support num_splits.
	const int block_n = D <= 64 ? 256 : (D <= 128 ? 128 : 64);
	const int num_n_blocks = (C + block_n - 1) / block_n;
	// Technically kBlockM = 64 only for the splitKV kernels, not the standard kernel.
	// In any case we don't expect seqlen_q to be larger than 64 for inference.
	const int num_m_blocks = (R + 64 - 1) / 64;
	const ccv_nnc_cuda_device_prop_t props = ccv_nnc_gpu_device_props();
	params.num_splits = num_splits_heuristic(batch_size * Hq * num_m_blocks, props.multi_processor_count, num_n_blocks, 128);
	if (params.num_splits > 1)
	{
		float* const workspace = (float*)ccv_nnc_stream_context_get_workspace(stream_context, (batch_size * Hq * R + params.num_splits * batch_size * Hq * R + params.num_splits * batch_size * Hq * R * params.d_rounded) * sizeof(float), CCV_TENSOR_GPU_MEMORY);
		params.softmax_lse_ptr = workspace;
		params.softmax_lseaccum_ptr = workspace + batch_size * Hq * R;
		params.oaccum_ptr = workspace + batch_size * Hq * R + params.num_splits * batch_size * Hq * R;
	} else {
		void* const workspace = ccv_nnc_stream_context_get_workspace(stream_context, batch_size * Hq * R * sizeof(float), CCV_TENSOR_GPU_MEMORY);
		params.softmax_lse_ptr = workspace;
	}
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	run_mha_fwd(params, stream, false);
	CUDA_ENFORCE(hipGetLastError());

	return CCV_NNC_EXEC_SUCCESS;
}

static int _ccv_nnc_scaled_dot_product_attention_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	return CCV_NNC_EXEC_INVALID;
}

#endif

REGISTER_COMMAND_BACKEND(CCV_NNC_SCALED_DOT_PRODUCT_ATTENTION_FORWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDA_SM80
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC;
	registry->tensor_datatypes = CCV_16F | CCV_QX;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_scaled_dot_product_attention_forw;
#endif
}

REGISTER_COMMAND_BACKEND(CCV_NNC_SCALED_DOT_PRODUCT_ATTENTION_BACKWARD, CCV_NNC_BACKEND_GPU_REF)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDA_SM80
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC;
	registry->tensor_datatypes = CCV_16F | CCV_QX;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_scaled_dot_product_attention_back;
#endif
}
