#include "hip/hip_runtime.h"
extern "C" {
#include <ccv.h>
#include <ccv_internal.h>
#include <nnc/ccv_nnc.h>
#include <nnc/ccv_nnc_easy.h>
#include <nnc/ccv_nnc_internal.h>
}
#include <nnc/gpu/ccv_nnc_compat.h>

#ifdef HAVE_CUDNN

template<typename NUM>
__global__ void _ccv_nnc_inv_std_kernel(const int count, const float epsilon, const NUM* const a, NUM* const b)
{
	CUDA_1D_KERNEL_LOOP(i, count) {
		b[i] = (NUM)(1. / sqrt((float)a[i] * (float)a[i] + epsilon));
	}
}

static int _ccv_nnc_rmsnorm_forw(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 2);
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	hipStream_t stream = ccv_nnc_stream_context_get_stream(stream_context);
	static const float one = 1, zero = 0;
	assert(output_size == 2);
	const ccv_nnc_cudnn_tensor_view_descriptor_t a = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)inputs[0]);
	assert(CCV_IS_TENSOR_CONTIGUOUS(inputs[1]));
	const ccv_nnc_cudnn_tensor_view_descriptor_t scale = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)inputs[1]);
	const ccv_nnc_cudnn_tensor_view_descriptor_t b = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)outputs[0]);
	assert(CCV_IS_TENSOR_CONTIGUOUS(outputs[1]));
	const int saved_datatype = outputs[1]->info.datatype;
	const ccv_nnc_cudnn_tensor_view_descriptor_t saved_inv_std = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)outputs[1]);
	int adim[CCV_NNC_MAX_DIM_ALLOC];
	int rdim[CCV_NNC_MAX_DIM_ALLOC];
	ccv_nnc_tensor_view_get_dim((ccv_nnc_tensor_view_t*)inputs[0], adim);
	ccv_nnc_tensor_view_get_dim((ccv_nnc_tensor_view_t*)outputs[1], rdim);
	assert(ccv_nnc_tensor_view_check_dim((ccv_nnc_tensor_view_t*)outputs[0], adim));
	int x;
	int n = 1;
	for (x = 0; x < CCV_NNC_MAX_DIM + 2; x++)
		n *= adim[x];
	for (x = 0; x < CCV_NNC_MAX_DIM + 2; x++)
		n /= rdim[x];
	int rcount = 1;
	for (x = 0; x < CCV_NNC_MAX_DIM + 2; x++)
		rcount *= rdim[x];
	const float inv_n = 1. / n;
	hipdnnReduceTensorDescriptor_t reduce = ccv_nnc_stream_context_get_reduce_tensor_descriptor(stream_context);
	size_t saved_inv_std_workspace_size = 0;
	hipdnnSetReduceTensorDescriptor(reduce, HIPDNN_REDUCE_TENSOR_NORM2, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN, HIPDNN_REDUCE_TENSOR_NO_INDICES, HIPDNN_32BIT_INDICES);
	CUDNN_ENFORCE(hipdnnGetReductionWorkspaceSize(cudnn, reduce, a.descriptor, saved_inv_std.descriptor, &saved_inv_std_workspace_size));
	const size_t workspace_size = saved_inv_std_workspace_size + sizeof(float) * rcount;
	uint8_t* const workspace = (uint8_t*)ccv_nnc_stream_context_get_workspace(stream_context, workspace_size, CCV_TENSOR_GPU_MEMORY);
	const float inv_n_sqrt = sqrt(inv_n);
	CUDNN_ENFORCE(hipdnnReduceTensor(cudnn, reduce, 0, 0, workspace, workspace_size, &inv_n_sqrt, a.descriptor, a.data.u8, &zero, saved_inv_std.descriptor, saved_inv_std.data.u8));
	// The epsilon is used a little bit differently from batch norm, it is inside the sqrt.
	const float epsilon = cmd.info.lnorm.epsilon;
	if (saved_datatype == CCV_32F)
		_ccv_nnc_inv_std_kernel<<<CUDA_GET_BLOCKS(rcount), CUDA_NUM_THREADS, 0, stream>>>(rcount, epsilon, saved_inv_std.data.f32, saved_inv_std.data.f32);
	else if (saved_datatype == CCV_16F)
		_ccv_nnc_inv_std_kernel<<<CUDA_GET_BLOCKS(rcount), CUDA_NUM_THREADS, 0, stream>>>(rcount, epsilon, (__half*)saved_inv_std.data.f16, (__half*)saved_inv_std.data.f16);
	hipdnnOpTensorDescriptor_t op = ccv_nnc_stream_context_get_op_tensor_descriptor(stream_context);
	hipdnnSetOpTensorDescriptor(op, HIPDNN_OP_TENSOR_MUL, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN);
	CUDNN_ENFORCE(hipdnnOpTensor(cudnn, op, &one, a.descriptor, a.data.u8, &one, saved_inv_std.descriptor, saved_inv_std.data.u8, &zero, b.descriptor, b.data.u8));
	CUDNN_ENFORCE(hipdnnOpTensor(cudnn, op, &one, b.descriptor, b.data.u8, &one, scale.descriptor, scale.data.u8, &zero, b.descriptor, b.data.u8));
	ccv_nnc_stream_context_return_reduce_tensor_descriptor(stream_context, reduce);
	ccv_nnc_stream_context_return_op_tensor_descriptor(stream_context, op);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(a);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(b);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(scale);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(saved_inv_std);
	return CCV_NNC_EXEC_SUCCESS;
}

static int _ccv_nnc_rmsnorm_back(const ccv_nnc_cmd_t cmd, const ccv_nnc_hint_t hint, const int flags, ccv_nnc_tensor_t* const* const inputs, const int input_size, ccv_nnc_tensor_t* const* const outputs, const int output_size, ccv_nnc_stream_context_t* const stream_context)
{
	assert(input_size == 6);
	assert(output_size >= 1);
	hipdnnHandle_t cudnn = ccv_nnc_stream_context_get_cudnn(stream_context);
	const ccv_nnc_cudnn_tensor_view_descriptor_t g = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)inputs[0]);
	const ccv_nnc_cudnn_tensor_view_descriptor_t a = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)inputs[2]);
	const ccv_nnc_cudnn_tensor_view_descriptor_t h = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)outputs[0]);
	assert(CCV_IS_TENSOR_CONTIGUOUS(inputs[3]));
	const ccv_nnc_cudnn_tensor_view_descriptor_t scale = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)inputs[3]);
	assert(CCV_IS_TENSOR_CONTIGUOUS(inputs[5]));
	const ccv_nnc_cudnn_tensor_view_descriptor_t saved_inv_std = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)inputs[5]);
	if (output_size > 1 && outputs[1])
		{ assert(CCV_IS_TENSOR_CONTIGUOUS(outputs[1])); }
	const ccv_nnc_cudnn_tensor_view_descriptor_t dscale = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, output_size > 1 ? (const ccv_nnc_tensor_view_t*)outputs[1] : 0);
	int gdim[CCV_NNC_MAX_DIM_ALLOC];
	int rdim[CCV_NNC_MAX_DIM_ALLOC];
	ccv_nnc_tensor_view_get_dim((ccv_nnc_tensor_view_t*)inputs[0], gdim);
	ccv_nnc_tensor_view_get_dim((ccv_nnc_tensor_view_t*)inputs[5], rdim);
	assert(ccv_nnc_tensor_view_check_dim((ccv_nnc_tensor_view_t*)inputs[2], gdim));
	assert(ccv_nnc_tensor_view_check_dim((ccv_nnc_tensor_view_t*)outputs[0], gdim));
	static const float one = 1, zero = 0;
	int x;
	int n = 1;
	for (x = 0; x < CCV_NNC_MAX_DIM + 2; x++)
		n *= gdim[x];
	for (x = 0; x < CCV_NNC_MAX_DIM + 2; x++)
		n /= rdim[x];
	int gcount = 1, rcount = 1;
	for (x = 0; x < CCV_NNC_MAX_DIM + 2; x++)
		gcount *= gdim[x], rcount *= rdim[x];
	const float neg_inv_n = -1. / n;
	hipdnnReduceTensorDescriptor_t reduce = ccv_nnc_stream_context_get_reduce_tensor_descriptor(stream_context);
	hipdnnSetReduceTensorDescriptor(reduce, HIPDNN_REDUCE_TENSOR_ADD, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN, HIPDNN_REDUCE_TENSOR_NO_INDICES, HIPDNN_32BIT_INDICES);
	size_t scale_workspace_size = 0;
	if (dscale.descriptor)
		{ CUDNN_ENFORCE(hipdnnGetReductionWorkspaceSize(cudnn, reduce, g.descriptor, dscale.descriptor, &scale_workspace_size)); }
	size_t inv_std_workspace_size = 0;
	CUDNN_ENFORCE(hipdnnGetReductionWorkspaceSize(cudnn, reduce, g.descriptor, saved_inv_std.descriptor, &inv_std_workspace_size));
	const size_t workspace_size = ccv_max(scale_workspace_size, inv_std_workspace_size);
	uint8_t* const workspace = (uint8_t*)ccv_nnc_stream_context_get_workspace(stream_context, workspace_size + sizeof(float) * gcount * 3 + sizeof(float) * rcount, CCV_TENSOR_GPU_MEMORY);
	float* const ahp = (float*)(workspace + workspace_size);
	const ccv_nnc_tensor_t aht = ccv_nnc_tensor(ahp, inputs[0]->info, 0);
	const ccv_nnc_cudnn_tensor_view_descriptor_t ah = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)&aht);
	float* const gssp = ahp + gcount;
	const ccv_nnc_tensor_t gsst = ccv_nnc_tensor(gssp, inputs[0]->info, 0);
	const ccv_nnc_cudnn_tensor_view_descriptor_t gss = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)&gsst);
	float* const ahgssp = gssp + gcount;
	const ccv_nnc_tensor_t ahgsst = ccv_nnc_tensor(ahgssp, inputs[0]->info, 0);
	const ccv_nnc_cudnn_tensor_view_descriptor_t ahgss = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)&ahgsst);
	float* const ahgssrp = ahgssp + gcount;
	const ccv_nnc_tensor_t ahgssrt = ccv_nnc_tensor(ahgssrp, inputs[5]->info, 0);
	const ccv_nnc_cudnn_tensor_view_descriptor_t ahgssr = ccv_nnc_cudnn_get_tensor_view_descriptor_for_op(stream_context, (const ccv_nnc_tensor_view_t*)&ahgssrt);
	hipdnnOpTensorDescriptor_t op = ccv_nnc_stream_context_get_op_tensor_descriptor(stream_context);
	hipdnnSetOpTensorDescriptor(op, HIPDNN_OP_TENSOR_MUL, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN);
	CUDNN_ENFORCE(hipdnnOpTensor(cudnn, op, &one, a.descriptor, a.data.u8, &one, saved_inv_std.descriptor, saved_inv_std.data.u8, &zero, ah.descriptor, ah.data.u8));
	CUDNN_ENFORCE(hipdnnOpTensor(cudnn, op, &one, ah.descriptor, ah.data.u8, &one, g.descriptor, g.data.u8, &zero, ahgss.descriptor, ahgss.data.u8));
	if (dscale.descriptor)
		{ CUDNN_ENFORCE(hipdnnReduceTensor(cudnn, reduce, 0, 0, workspace, workspace_size, &one, ahgss.descriptor, ahgss.data.u8, &zero, dscale.descriptor, dscale.data.u8)); }
	CUDNN_ENFORCE(hipdnnOpTensor(cudnn, op, &one, g.descriptor, g.data.u8, &one, scale.descriptor, scale.data.u8, &zero, gss.descriptor, gss.data.u8));
	CUDNN_ENFORCE(hipdnnOpTensor(cudnn, op, &one, gss.descriptor, gss.data.u8, &one, saved_inv_std.descriptor, saved_inv_std.data.u8, &zero, gss.descriptor, gss.data.u8));
	CUDNN_ENFORCE(hipdnnOpTensor(cudnn, op, &one, ah.descriptor, ah.data.u8, &one, gss.descriptor, gss.data.u8, &zero, ahgss.descriptor, ahgss.data.u8));
	CUDNN_ENFORCE(hipdnnReduceTensor(cudnn, reduce, 0, 0, workspace, workspace_size, &one, ahgss.descriptor, ahgss.data.u8, &zero, ahgssr.descriptor, ahgssr.data.u8));
	ccv_nnc_stream_context_return_reduce_tensor_descriptor(stream_context, reduce);
	CUDNN_ENFORCE(hipdnnOpTensor(cudnn, op, &one, ah.descriptor, ah.data.u8, &one, ahgssr.descriptor, ahgssr.data.u8, &zero, ah.descriptor, ah.data.u8));
	hipdnnSetOpTensorDescriptor(op, HIPDNN_OP_TENSOR_ADD, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN);
	CUDNN_ENFORCE(hipdnnOpTensor(cudnn, op, &one, gss.descriptor, gss.data.u8, &neg_inv_n, ah.descriptor, ah.data.u8, &zero, h.descriptor, h.data.u8));
	ccv_nnc_stream_context_return_op_tensor_descriptor(stream_context, op);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(g);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(a);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(h);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(scale);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(saved_inv_std);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(dscale);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(ah);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(gss);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(ahgss);
	ccv_nnc_cudnn_deinit_tensor_view_descriptor(ahgssr);
	return CCV_NNC_EXEC_SUCCESS;
}

#endif

REGISTER_COMMAND_BACKEND(CCV_NNC_RMSNORM_FORWARD, CCV_NNC_BACKEND_GPU_CUDNN)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDNN
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_rmsnorm_forw;
#endif
}

REGISTER_COMMAND_BACKEND(CCV_NNC_RMSNORM_BACKWARD, CCV_NNC_BACKEND_GPU_CUDNN)(ccv_nnc_cmd_backend_registry_t* const registry)
{
#ifdef HAVE_CUDNN
	registry->tensor_formats = CCV_TENSOR_FORMAT_NCHW | CCV_TENSOR_FORMAT_NHWC | CCV_TENSOR_FORMAT_CHWN;
	registry->tensor_datatypes = CCV_32F | CCV_16F;
	registry->tensor_memory = CCV_TENSOR_GPU_MEMORY;
	registry->algorithms = 1;
	registry->exec = _ccv_nnc_rmsnorm_back;
#endif
}
